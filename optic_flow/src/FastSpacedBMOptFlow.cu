#include "hip/hip_runtime.h"
#include "optic_flow/FastSpacedBMOptFlow.h"

#include <iostream>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <opencv2/gpu/gpumat.hpp>

#define arraySize 20

static inline void _safe_cuda_call(hipError_t err, const char* msg, const char* file_name, const int line_number)
{
    if(err!=hipSuccess)
    {
        std::fprintf(stderr,"%s\n\nFile: %s\n\nLine Number: %d\n\nReason: %s\n",
                msg,file_name,line_number,hipGetErrorString(err));
        std::cin.get();
        exit(EXIT_FAILURE);
    }
}

#define SAFE_CALL(call,msg) _safe_cuda_call((call),(msg),__FILE__,__LINE__)

__global__ void _FastSpacedBMOptFlow_kernel(const cv::gpu::PtrStepSzb input_1,
                                            const cv::gpu::PtrStepSzb input_2,
                                            cv::gpu::PtrStepSz<signed char> output_X,
                                            cv::gpu::PtrStepSz<signed char> output_Y,
                                            int blockSize,
                                            int blockStep,
                                            int scanRadius,
                                            int width,
                                            int height)
{
    int scanDiameter = scanRadius*2+1;
    __shared__ int abssum[arraySize][arraySize];


    if( ((blockIdx.x+1) * (blockSize + 1+scanRadius*2) < width)
            &&
            ((blockIdx.y+1) * (blockSize + 1+scanRadius*2)) < height )
    {

            for (int i=0;i<blockSize;i++)
            {
                for (int j=0;j<blockSize;j++)
                {
                    abssum[threadIdx.y][threadIdx.x]+=
                            (abs(input_1((blockIdx.y + scanRadius + i),(blockIdx.x + scanRadius + j))
                            -
                            input_2((blockIdx.y + i + threadIdx.y),
                                    (blockIdx.x + j + threadIdx.x))));
                }

            }

            __syncthreads();
            __shared__ int minval[arraySize];
            __shared__ signed char minX[arraySize];
            signed char minY;

            if (threadIdx.y == 0)
            {
                minval[threadIdx.x] = abssum[threadIdx.x][0];
                minX[threadIdx.x] = -scanRadius;
                for (int i=1;i<scanDiameter;i++)
                {
                    if (minval[threadIdx.x] > abssum[threadIdx.x][i])
                    {
                        minval[threadIdx.x] = abssum[threadIdx.x][i];
                        minX[threadIdx.x] = threadIdx.x-scanRadius;
                    }
                }
            }
             __syncthreads();

            if ( (threadIdx.y == 0) && (threadIdx.x == 0))
            {
                int minvalFin = minval[0];
                minY = -scanRadius;
                for (int i=1;i<scanDiameter;i++)
                {
                    if (minvalFin > minval[i])
                    {
                        minvalFin = minval[i];
                        minY = i-scanRadius;
                    }
                }
                output_Y(blockIdx.y,blockIdx.x) = minY;
                output_X(blockIdx.y,blockIdx.x) = minX[minY+scanRadius];
            }



    }





}

__global__ void _HistogramMaximum(const cv::gpu::PtrStepSz<signed char> input,
                                  int scanRadius,
                                  signed char *value)
{

    __shared__ int Histogram[arraySize];

    if ((threadIdx.x < arraySize) && (threadIdx.y == 0))
        Histogram[threadIdx.x] = 0;


    __syncthreads();
    Histogram[input(threadIdx.y,threadIdx.x)+scanRadius]++;
    __syncthreads();


    if ( (threadIdx.y == 0) && (threadIdx.x == 0))
    {
        int MaxIndex = 0;
        char  MaxVal = 0;
        for (int i=0;i<blockDim.y;i++)
        {
            if (MaxVal < Histogram[i])
            {
                MaxVal = Histogram[i];
                MaxIndex = i;
            }
        }
        *value = MaxIndex - scanRadius;
    }

}

__global__ void _CopyMatrix(const cv::gpu::PtrStepSzb input,
                            cv::gpu::PtrStepb output,
                            int blockSize,
                            int width,
                            int height) //test for CUDA CV basics
{
        output(threadIdx.y,threadIdx.x) = input(threadIdx.y,threadIdx.x);
}

void ResetCudaDevice()
{

    SAFE_CALL(hipDeviceReset(),"Killing previous kernels Failed!");
}

void FastSpacedBMOptFlow(cv::InputArray _imPrev, cv::InputArray _imCurr,
                         cv::OutputArray _imOutX, cv::OutputArray _imOutY,
                         int blockSize,
                         int blockStep,
                         int scanRadius,
                         signed char &outX,
                         signed char &outY)
{
    const cv::gpu::GpuMat imPrev = _imPrev.getGpuMat();
    const cv::gpu::GpuMat imCurr = _imCurr.getGpuMat();
    if (imPrev.size() != imCurr.size())
    {
        std::fprintf(stderr,"Input images do not match in sizes!");
        std::cin.get();
        exit(EXIT_FAILURE);
    }
    if ((imPrev.type() != CV_8UC1) || (imCurr.type() != CV_8UC1))
    {
        std::fprintf(stderr,"Input image/s are not of the CV_8UC1 type!");
        std::cin.get();
        exit(EXIT_FAILURE);
    }

    int scanDiameter = 2*scanRadius+1;
    int blockszX = scanDiameter+blockSize;
    int blockszY = scanDiameter+blockSize;

    const dim3 block(scanDiameter, scanDiameter);
    const dim3 grid((imPrev.cols)/blockszX, (imPrev.rows)/blockszY);

    _imOutX.create(grid.x,grid.y,CV_8UC1);
    const cv::gpu::GpuMat imOutX = _imOutX.getGpuMat();
    _imOutY.create(grid.x,grid.y,CV_8UC1);
    const cv::gpu::GpuMat imOutY = _imOutY.getGpuMat();

    //_CopyMatrix<<<1,blockM>>>(imPrev,imOutX,blockSize,imPrev.cols,imPrev.rows);
    _FastSpacedBMOptFlow_kernel<<<grid,block>>>(imPrev,imCurr,imOutX,imOutY,
                                                blockSize,blockStep,scanRadius,
                                                imCurr.cols, imCurr.rows);

    signed char outX_l;
    signed char outY_l;

    signed char* outX_g;
    signed char* outY_g;
    hipMalloc(&outX_g, sizeof(signed char));
    hipMalloc(&outY_g, sizeof(signed char));

    _HistogramMaximum<<<1,grid>>>(imOutX,scanRadius,outX_g);
    _HistogramMaximum<<<1,grid>>>(imOutY,scanRadius,outY_g);

    SAFE_CALL(hipMemcpy(&outX_l,outX_g,sizeof(signed char),hipMemcpyDeviceToHost),"Memcpy to host failed");
    SAFE_CALL(hipMemcpy(&outY_l,outY_g,sizeof(signed char),hipMemcpyDeviceToHost),"Memcpy to host failed");

    hipFree(outX_g);
    hipFree(outY_g);

   SAFE_CALL(hipDeviceSynchronize(),"Kernel Launch Failed");

   outX = outX_l;
   outY = outY_l;



}
