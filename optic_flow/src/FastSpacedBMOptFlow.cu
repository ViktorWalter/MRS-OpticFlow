#include "hip/hip_runtime.h"
#include "optic_flow/FastSpacedBMOptFlow.h"

#include <iostream>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <opencv2/gpu/gpumat.hpp>

#define arraySize 50
#define MinValThreshold (scanRadius*scanRadius*0.2)

static inline void _safe_cuda_call(hipError_t err, const char* msg, const char* file_name, const int line_number)
{
    if(err!=hipSuccess)
    {
        std::fprintf(stderr,"%s\n\nFile: %s\n\nLine Number: %d\n\nReason: %s\n",
                msg,file_name,line_number,hipGetErrorString(err));
        std::cin.get();
        exit(EXIT_FAILURE);
    }
}

#define SAFE_CALL(call,msg) _safe_cuda_call((call),(msg),__FILE__,__LINE__)

__global__ void _FastSpacedBMOptFlow_kernel(const cv::gpu::PtrStepSzb input_1,
                                            const cv::gpu::PtrStepSzb input_2,
                                            cv::gpu::PtrStepSz<signed char> output_X,
                                            cv::gpu::PtrStepSz<signed char> output_Y,
                                            int blockSize,
                                            int blockStep,
                                            int scanRadius,
                                            int width,
                                            int height)
{


    int scanDiameter = scanRadius*2+1;
    __shared__ int abssum[arraySize][arraySize];


        abssum[threadIdx.y][threadIdx.x] = 0;

            for (int i=0;i<blockSize;i++)
            {
                for (int j=0;j<blockSize;j++)
                {
                    atomicAdd(&(abssum[threadIdx.y][threadIdx.x]),
                            abs(
                                 input_1(((blockIdx.y*(blockSize+blockStep)) + scanRadius + i),
                                         ((blockIdx.x*(blockSize+blockStep)) + scanRadius + j))
                                 -
                                 input_2(((blockIdx.y*(blockSize+blockStep)) + i + threadIdx.y),
                                         ((blockIdx.x*(blockSize+blockStep)) + j + threadIdx.x)))
                            );
                }

            }

            __syncthreads();

            __shared__ int minval[arraySize];
            __shared__ signed char minX[arraySize];
            signed char minY;

            if (threadIdx.y == 0)
            {
                minval[threadIdx.x] = abssum[threadIdx.x][0];
                minX[threadIdx.x] = -scanRadius;
                for (int i=1;i<scanDiameter;i++)
                {
                    if (minval[threadIdx.x] > abssum[threadIdx.x][i])
                    {
                        minval[threadIdx.x] = abssum[threadIdx.x][i];
                        minX[threadIdx.x] = i-scanRadius;
                    }
                }
            }
             __syncthreads();



            if ( (threadIdx.y == 0) && (threadIdx.x == 0))
            {
                int minvalFin = minval[0];
                minY = -scanRadius;
                for (int i=1;i<scanDiameter;i++)
                {
                    if (minvalFin > minval[i])
                    {
                        minvalFin = minval[i];
                        minY = i-scanRadius;
                    }
                }
                output_Y(blockIdx.y,blockIdx.x) = minY;
                output_X(blockIdx.y,blockIdx.x) = minX[minY+scanRadius];

                if ((abssum[scanRadius][scanRadius] - minvalFin) <= MinValThreshold)  //if the difference is small, then it is considered to be noise in a uniformly colored area
                {
                    output_Y(blockIdx.y,blockIdx.x) = 0;
                    output_X(blockIdx.y,blockIdx.x) = 0;
                }


            }


}

__global__ void _HistogramMaximum(const cv::gpu::PtrStepSz<signed char> input,
                                  int scanRadius,
                                  int scanDiameter,
                                  signed char *value)
{

    __shared__ int Histogram[arraySize];


    int index = (threadIdx.y*blockDim.x+threadIdx.x);
    if (index < scanDiameter)
        Histogram[index] = 0;

    __syncthreads();

    atomicAdd(&(Histogram[input(threadIdx.y,threadIdx.x)+scanRadius]),1);

    __syncthreads();


    if ( (threadIdx.y == 0) && (threadIdx.x == 0))
    {
        int MaxIndex = 0;
        char  MaxVal = 0;

        for (int i=0;i<scanDiameter;i++)
        {
            if (MaxVal < Histogram[i])
            {
                MaxVal = Histogram[i];
                MaxIndex = i;
            }
        }
        *value = MaxIndex - scanRadius;


    }

}

__global__ void _CopyMatrix(const cv::gpu::PtrStepSzb input,
                            cv::gpu::PtrStepb output,
                            int blockSize,
                            int width,
                            int height) //test for CUDA CV basics
{
        output(threadIdx.y,threadIdx.x) = input(threadIdx.y,threadIdx.x);
}

void ResetCudaDevice()
{

    SAFE_CALL(hipDeviceReset(),"Killing previous kernels Failed!");
}

void FastSpacedBMOptFlow(cv::InputArray _imPrev, cv::InputArray _imCurr,
                         cv::OutputArray _imOutX, cv::OutputArray _imOutY,
                         int blockSize,
                         int blockStep,
                         int scanRadius,
                         signed char &outX,
                         signed char &outY)
{
    const cv::gpu::GpuMat imPrev = _imPrev.getGpuMat();
    const cv::gpu::GpuMat imCurr = _imCurr.getGpuMat();
    if (imPrev.size() != imCurr.size())
    {
        std::fprintf(stderr,"Input images do not match in sizes!");
        std::cin.get();
        exit(EXIT_FAILURE);
    }
    if ((imPrev.type() != CV_8UC1) || (imCurr.type() != CV_8UC1))
    {
        std::fprintf(stderr,"Input image/s are not of the CV_8UC1 type!");
        std::cin.get();
        exit(EXIT_FAILURE);
    }

    int scanDiameter = (2*scanRadius)+1;
    int blockszX = blockStep+blockSize;
    int blockszY = blockStep+blockSize;

    const dim3 block(scanDiameter, scanDiameter);
    const dim3 grid((imPrev.cols-scanRadius*2)/blockszX,(imPrev.rows-scanRadius*2)/blockszY);
    //const dim3 grid(1,1);

    _imOutX.create(grid.x,grid.y,CV_8SC1);
    const cv::gpu::GpuMat imOutX = _imOutX.getGpuMat();
    _imOutY.create(grid.x,grid.y,CV_8SC1);
    const cv::gpu::GpuMat imOutY = _imOutY.getGpuMat();

    //_CopyMatrix<<<1,blockM>>>(imPrev,imOutX,blockSize,imPrev.cols,imPrev.rows);
    _FastSpacedBMOptFlow_kernel<<<grid,block,0>>>(imPrev,imCurr,imOutX,imOutY,
                                                blockSize,blockStep,scanRadius,
                                                imCurr.cols, imCurr.rows);

    signed char outX_l;
    signed char outY_l;

    signed char* outX_g;
    signed char* outY_g;
    hipMalloc(&outX_g, sizeof(signed char));
    hipMalloc(&outY_g, sizeof(signed char));

    SAFE_CALL(hipDeviceSynchronize(),"Kernel Launch Failed 1");
    _HistogramMaximum<<<1,grid,1>>>(imOutX,scanRadius, scanDiameter,outX_g);
    _HistogramMaximum<<<1,grid,1>>>(imOutY,scanRadius, scanDiameter,outY_g);

    SAFE_CALL(hipDeviceSynchronize(),"Kernel Launch Failed 2");

    SAFE_CALL(hipMemcpy(&outX_l,outX_g,sizeof(signed char),hipMemcpyDeviceToHost),"Memcpy to host failed");
    SAFE_CALL(hipMemcpy(&outY_l,outY_g,sizeof(signed char),hipMemcpyDeviceToHost),"Memcpy to host failed");

    hipFree(outX_g);
    hipFree(outY_g);

   SAFE_CALL(hipDeviceSynchronize(),"Kernel Launch Failed 3");

   outX = outX_l;
   outY = outY_l;

}
