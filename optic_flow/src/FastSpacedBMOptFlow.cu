#include "hip/hip_runtime.h"
#include "optic_flow/FastSpacedBMOptFlow.h"

#include <iostream>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <opencv2/gpu/gpumat.hpp>

#define arraySize 20

static inline void _safe_cuda_call(hipError_t err, const char* msg, const char* file_name, const int line_number)
{
    if(err!=hipSuccess)
    {
        std::fprintf(stderr,"%s\n\nFile: %s\n\nLine Number: %d\n\nReason: %s\n",
                msg,file_name,line_number,hipGetErrorString(err));
        std::cin.get();
        exit(EXIT_FAILURE);
    }
}

#define SAFE_CALL(call,msg) _safe_cuda_call((call),(msg),__FILE__,__LINE__)

__global__ void _FastSpacedBMOptFlow_kernel(unsigned char* input_1,
                                    unsigned char* input_2,
                                    signed char* output_X,
                                    signed char* output_Y,
                                    int blockSize,
                                    int blockStep,
                                    int scanRadius,
                                    int width,
                                    int height)
{
    int scanDiameter = scanRadius*2+1;
    __shared__ int abssum[arraySize][arraySize];

    if( (blockIdx.x * (blockSize + 1+scanRadius*2) < width) && (blockIdx.y * (blockSize + 1+scanRadius*2)) < height )
    {
            for (int i=0;i<blockSize;i++)
            {
                for (int j=0;j<blockSize;i++)
                {
                    abssum[threadIdx.y][threadIdx.x]+=
                            (abs(input_1[
                            (width * (blockIdx.y + i))    //y
                            + blockIdx.x + scanRadius + j] -        //x
                            input_2[
                            (width * (blockIdx.y + i + threadIdx.y - (blockSize/2))) //y
                            + blockIdx.x + j + threadIdx.x - (blockSize/2)]));      //x
                }

            }

            __syncthreads();
            __shared__ int minval[arraySize];
            char minX[arraySize];
            char minY;

            if (threadIdx.y == 0)
            {
                minval[threadIdx.x] = abssum[threadIdx.x][0];
                minX[threadIdx.x] = -scanRadius;
                for (int i=1;i<scanDiameter;i++)
                {
                    if (minval[threadIdx.x] > abssum[threadIdx.x][i])
                    {
                        minval[threadIdx.x] = abssum[threadIdx.x][i];
                        minX[threadIdx.x] = threadIdx.x-scanRadius;
                    }
                }
            }
             __syncthreads();

            if ( (threadIdx.y == 0) && (threadIdx.x == 0))
            {
                int minvalFin = minval[0];
                minY = -scanRadius;
                for (int i=1;i<scanDiameter;i++)
                {
                    if (minvalFin > minval[i])
                    {
                        minvalFin = minval[i];
                        minY = i-scanRadius;
                    }
                }
                output_Y[width*blockIdx.y + blockIdx.x] = minY;
                output_X[width*blockIdx.y + blockIdx.x] = minX[minY+scanRadius];
            }



    }





}

__global__ void _HistogramMaximum(signed char* input_1,
                                  int scanRadius,
                                  signed char* value)
{
    __shared__ int Histogram[arraySize];

    if ((threadIdx.x < arraySize) && (threadIdx.y == 0))
        Histogram[threadIdx.x] = 0;

    __syncthreads();
    Histogram[input_1[blockDim.y*threadIdx.y+threadIdx.x]+scanRadius]++;

    __syncthreads();

    if ( (threadIdx.y == 0) && (threadIdx.x == 0))
    {
        int MaxIndex = 0;
        char  MaxVal = 0;
        for (int i=1;i<blockDim.y;i++)
        {
            if (MaxVal < Histogram[i])
            {
                MaxVal = Histogram[i];
                MaxIndex = i;
            }
        }
        *value = MaxIndex - scanRadius;
    }

}

void ResetCudaDevice()
{

    SAFE_CALL(hipDeviceReset(),"Killing previous kernels Failed!");
}

void FastSpacedBMOptFlow(cv::gpu::GpuMat &imPrev, cv::gpu::GpuMat &imCurr,
                         cv::gpu::GpuMat &imOutX, cv::gpu::GpuMat &imOutY,
                         int blockSize,
                         int blockStep,
                         int scanRadius,
                         signed char &outX,
                         signed char &outY)
{

    if (imPrev.size() != imCurr.size())
    {
        std::fprintf(stderr,"Input images do not match in sizes!");
        std::cin.get();
        exit(EXIT_FAILURE);
    }
    if ((imPrev.type() != CV_8UC1) || (imCurr.type() != CV_8UC1))
    {
        std::fprintf(stderr,"Input image/s are not of the CV_8UC1 type!");
        std::cin.get();
        exit(EXIT_FAILURE);
    }

    int scanDiameter = 2*scanRadius+1;
    int blockszX = scanDiameter+blockSize;
    int blockszY = scanDiameter+blockSize;

    imOutX = cv::gpu::GpuMat((imPrev.cols)/blockszX, (imPrev.rows)/blockszY,
                             CV_8SC1);
    imOutY = cv::gpu::GpuMat((imPrev.cols)/blockszX, (imPrev.rows)/blockszY,
                             CV_8SC1);

    unsigned char* pi1 = (unsigned char*)imPrev.data;
    unsigned char* pi2 = (unsigned char*)imCurr.data;
    signed char* po1 = (signed char*)imOutX.data;
    signed char* po2 = (signed char*)imOutY.data;


    const dim3 block(scanDiameter, scanDiameter);
    const dim3 grid((imPrev.cols)/blockszX, (imPrev.rows)/blockszY);

    std::fprintf(stderr,"OptFlow Kernel:\n");

    _FastSpacedBMOptFlow_kernel<<<grid,block>>>(pi1,pi2,po1,po2,
                                                blockSize,blockStep,scanRadius,
                                                imCurr.cols, imCurr.rows);

    signed char* outX_l;
    signed char* outY_l;

    hipHostMalloc((void**)&outX_l,1);
    hipHostMalloc((void**)&outY_l,1);

    signed char* outX_g;
    signed char* outY_g;
    hipMalloc((void**)&outX_g, 1);
    hipMalloc((void**)&outY_g, 1);

    std::fprintf(stderr,"Histogram Kernel:\n");

    std::fprintf(stderr,"X:\n");
    _HistogramMaximum<<<1,block>>>(po1,scanRadius,outX_g);
    std::fprintf(stderr,"Y:\n");
    _HistogramMaximum<<<1,block>>>(po2,scanRadius,outY_g);


    std::fprintf(stderr,"Copying to Memory:\n");
    memcpy(outX_l,outX_g,1);
    memcpy(outY_l,outY_g,1);

    hipFree(outX_g);
    hipFree(outY_g);

    std::fprintf(stderr,"Synchronizing:\n");
   SAFE_CALL(hipDeviceSynchronize(),"Kernel Launch Failed");

   outX = *outX_l;
   outY = *outY_l;

   hipHostFree(outX_l);
   hipHostFree(outY_l);

   std::fprintf(stderr,"Kernel returning\n");


}
